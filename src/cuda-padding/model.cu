#include "hip/hip_runtime.h"
#include "model.h"

int hidden_dim = 300;

void print(size_t n, data_t *a) {
    data_t *temp = (data_t*)malloc(n * sizeof(data_t));
    hipMemcpy(temp, a, n * sizeof(data_t), hipMemcpyDeviceToHost);
    for(int i = 0; i < n; ++i) {
        printf("%f ", temp[i]);
    }
    printf("\n");
    free(temp);
}

void print2D(size_t n, size_t m, data_t *a, size_t padding) {
    data_t *temp = (data_t*)malloc(n * m * sizeof(data_t));
    hipMemcpy2D(temp, m * sizeof(data_t), a, padding * sizeof(data_t), m * sizeof(data_t), n, hipMemcpyDeviceToHost);
    for(int i = 0; i < n; ++i) {
        for(int j = 0; j < m; ++j) {
            printf("%f ", a[i * m + j]);
        }
        printf("\n");
    }
    free(temp);
}

void set_hidden_layer_size(int new_size) { 
    hidden_dim = new_size;
}

data_t *toDevice2D(const size_t n, const size_t m, data_t *x, size_t *padding) {
    data_t *ret;
    hipMallocPitch(&ret, padding, m * sizeof(data_t), n);
    hipMemcpy2D(ret, *padding, x, m * sizeof(data_t), m * sizeof(data_t), n, hipMemcpyHostToDevice);
    free(x); *padding /= sizeof(data_t);
    return ret;
}

data_t *toDevice(const size_t n, data_t *x) {
    data_t *ret;
    hipMalloc(&ret, n * sizeof(data_t));
    hipMemcpy(ret, x, n * sizeof(data_t), hipMemcpyHostToDevice);
    free(x); 
    return ret;
}

int *toDevice(const size_t n, int *x) {
    int *ret;
    hipMalloc(&ret, n * sizeof(int));
    hipMemcpy(ret, x, n * sizeof(int), hipMemcpyHostToDevice);
    free(x); 
    return ret;
}

data_t *toHost(const size_t n, data_t *x) {
    data_t *ret;
    ret = (data_t*)malloc(n * sizeof(data_t));
    hipMemcpy(ret, x, n * sizeof(data_t), hipMemcpyDeviceToHost);
    hipFree(x);
    return ret;
} 

int *toHost(const size_t n, int *x) {
    int *ret;
    ret = (int*)malloc(n * sizeof(int));
    hipMemcpy(ret, x, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(x);
    return ret;
} 

void fill_uniform(size_t n, data_t *a, const data_t L, const data_t R) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<data_t> dist(L, R);
    for (size_t i = 0; i < n; ++i) {
        a[i] = dist(gen);
    }
}

__global__ void __fill__(size_t n, data_t *a, const data_t val) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) a[i] = val;
}

void fill_val(size_t n, data_t *a, const data_t val) {
    int numBlocks = (n - 1) / 32 + 1;
    __fill__<<<numBlocks, 32>>>(n, a, val);
}

__global__ void __fill2D__(size_t n, size_t m, data_t *a, const data_t val, size_t padding) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i < n && j < m) {
        a[i * padding + j] = val;
    }
}

void fill_val2D(size_t n, size_t m, data_t *a, const data_t val, size_t padding) {
    dim3 numBlocks((n - 1) / 32 + 1, (m - 1) / 32 + 1);
    dim3 numThreads(32, 32);
    __fill2D__<<<numBlocks, numThreads>>>(n, m, a, val, padding);
}

data_t sigmoid(data_t x) {
    return 1.0 / (1.0 + exp(-x));
}

__device__ data_t __sigmoid__(data_t x) {
    return 1.0 / (1.0 + exp(-x));
}

__global__ void __accuracy__(const size_t batch_size, data_t *pred, int *label, int *correct, size_t padding) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < batch_size) {
        int argmax = -1;
        for(int j = 0; j < output_dim; ++j) {
            if(argmax == -1 || pred[i * padding + j] > pred[i * padding + argmax])
                argmax = j;
        }
        if(argmax == label[i]) {
            atomicAdd(correct, 1);
        }
    }
} 

data_t accuracy(const size_t batch_size, data_t *pred, int *label, size_t padding) {
    int correct = 0;
    int *device_correct;
    hipMalloc(&device_correct, sizeof(int));
    hipMemset(device_correct, 0, sizeof(int));
    int numBlocks = (batch_size - 1) / 32 + 1;
    __accuracy__<<<numBlocks, 32>>>(batch_size, pred, label, device_correct, padding);
    hipMemcpy(&correct, device_correct, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(device_correct);
    return (data_t)correct / batch_size;
}

// some functions

// c = a * b
__global__ void __mul__(size_t n, size_t m, size_t p, data_t *a, data_t *b, data_t *c, size_t padding_a, size_t padding_b, size_t padding_c) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if(i < n && j < p) {
        c[i * padding_c + j] = 0.0;
        for(int k = 0; k < m; ++k) {
            c[i * padding_c + j] += a[i * padding_a + k] * b[k * padding_b + j];
        }
    }
}

// b[i, j] += a[j]
__global__ void __add__(size_t n, size_t m, data_t *a, data_t *b, size_t padding_b) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
        b[i * padding_b + j] += a[j];
    }
}

// b[j] += a[i, j]
__global__ void __accumulate__(size_t n, size_t m, data_t *a, data_t *b, size_t padding_a) {
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    if(j < m) {
        for(int i = 0; i < n; ++i) {
            b[j] += a[i * padding_a + j];
        }
    }
}

// a = sigmoid(a) and accumulate its gradient
__global__ void __sigmoid__(size_t n, size_t m, data_t *a, data_t *grad, size_t padding_a) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
        a[i * padding_a + j] = __sigmoid__(a[i * padding_a + j]);
        atomicAdd(grad + j, a[i * padding_a + j] * (1.0 - a[i * padding_a + j]));
    }
}

__global__ void __div__(size_t n, data_t *a, data_t val) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < n) {
        a[i] /= val;
    }
}

// c[i] = a[i] * b[i]
__global__ void __mul__(size_t n, data_t *a, data_t *b, data_t *c) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < n) {
        c[i] = a[i] * b[i];
    }
}

// b = a;
__global__ void __copy__(size_t n, data_t *a, data_t *b) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < n) {
        b[i] = a[i];
    }
}

__global__ void __copy2D__(size_t n, size_t m, data_t *a, data_t *b, size_t padding_a, size_t padding_b) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
        b[i * padding_b + j] = a[i * padding_a + j];
    }
}

// c = dot(a, b)
__global__ void __dot__(size_t n, size_t m, data_t *a, data_t *b, data_t *c, size_t padding_c) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
        c[i * padding_c + j] = a[i] * b[j];
    }
}

// c[i] = sum(a[i, j] * b[j])
__global__ void __sum__(size_t n, size_t m, data_t *a, data_t *b, data_t *c, size_t padding_a) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < n) {
        c[i] = 0.0;
        for(int j = 0; j < m; ++j) {
            c[i] += a[i * padding_a + j] * b[j];
        }
    }
}

// b -= ratio * a;
__global__ void __sub_eq__(size_t n, data_t *a, data_t *b, data_t ratio) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;      
    if(i < n) {
        b[i] -= a[i] * ratio;
    }
}

__global__ void __sub_eq_2D__(size_t n, size_t m, data_t *a, data_t *b, data_t ratio, size_t padding_a, size_t padding_b) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
        b[i * padding_b + j] = a[i * padding_a + j];
    }
}

// model

model::~model()  {
    hipFree(w_1);
    hipFree(w_2);
    hipFree(sum_1);
    hipFree(sum_2);
    hipFree(bias_1);
    hipFree(bias_2);
    hipFree(grad_w_1);
    hipFree(grad_w_2);
    hipFree(grad_loss);
    hipFree(prev_grad);
    hipFree(grad_bias_1);
    hipFree(grad_bias_2);
    hipFree(grad_sigmoid_1);
    hipFree(grad_sigmoid_2);
}

model::model() {
    data_t *temp;
    size_t pitch;
    // init first layer bias
    bias_1 = (data_t*)malloc(hidden_dim * sizeof(data_t));
    fill_uniform(hidden_dim, bias_1, -sqrt(input_dim), sqrt(input_dim));
    hipMallocPitch(&temp, &pitch, hidden_dim * sizeof(data_t), 1);
    hipMemcpy(temp, bias_1, hidden_dim * sizeof(data_t), hipMemcpyHostToDevice);
    free(bias_1); bias_1 = temp;
    // init first layer weight
    w_1 = (data_t*)malloc(input_dim * hidden_dim * sizeof(data_t));
    fill_uniform(input_dim * hidden_dim, w_1, -sqrt(input_dim), sqrt(input_dim));
    hipMallocPitch(&temp, &padding_w_1, hidden_dim * sizeof(data_t), input_dim);
    hipMemcpy2D(temp, padding_w_1, w_1, hidden_dim * sizeof(data_t), hidden_dim * sizeof(data_t), input_dim, hipMemcpyHostToDevice);
    free(w_1); w_1 = temp;
    // init second layer bias
    bias_2 = (data_t*)malloc(output_dim * sizeof(data_t));
    fill_uniform(output_dim, bias_2, -sqrt(hidden_dim), sqrt(hidden_dim));
    hipMallocPitch(&temp, &pitch, output_dim * sizeof(data_t), 1);
    hipMemcpy(temp, bias_2, output_dim * sizeof(data_t), hipMemcpyHostToDevice);
    free(bias_2); bias_2 = temp;
    // init second layer weight
    w_2 = (data_t*)malloc(hidden_dim * output_dim * sizeof(data_t));
    fill_uniform(hidden_dim * output_dim, w_2, -sqrt(hidden_dim), sqrt(hidden_dim));
    hipMallocPitch(&temp, &padding_w_2, output_dim * sizeof(data_t), hidden_dim);
    hipMemcpy2D(temp, padding_w_2, w_2, output_dim * sizeof(data_t), output_dim * sizeof(data_t), hidden_dim, hipMemcpyHostToDevice);
    free(w_2); w_2 = temp;
    // first linear layer
    hipMallocPitch(&sum_1, &pitch, input_dim * sizeof(data_t), 1);
    hipMallocPitch(&grad_sigmoid_1, &pitch, hidden_dim * sizeof(data_t), 1);
    hipMallocPitch(&grad_bias_1, &pitch, hidden_dim * sizeof(data_t), 1);
    hipMallocPitch(&grad_w_1, &padding_grad_w_1, hidden_dim * sizeof(data_t), input_dim);    
    // second linear layer
    hipMallocPitch(&sum_2, &pitch, hidden_dim * sizeof(data_t), 1);
    hipMallocPitch(&grad_sigmoid_2, &pitch, output_dim * sizeof(data_t), 1);
    hipMallocPitch(&grad_bias_2, &pitch, output_dim * sizeof(data_t), 1);
    hipMallocPitch(&grad_w_2, &padding_grad_w_2, output_dim * sizeof(data_t), hidden_dim);
    // loss 
    hipMallocPitch(&grad_loss, &pitch, output_dim * sizeof(data_t), 1);
    hipMallocPitch(&prev_grad, &pitch, hidden_dim * sizeof(data_t), 1);
    padding_grad_w_1 /= sizeof(data_t);
    padding_grad_w_2 /= sizeof(data_t);
    padding_w_1 /= sizeof(data_t);
    padding_w_2 /= sizeof(data_t);
}

data_t* model::forward(size_t batch_size, data_t *x, size_t padding_x, size_t *padding_pred) {
    batch_count += batch_size;
    // first layer forward
    data_t *temp_1; 
    size_t padding_temp_1;
    hipMallocPitch(&temp_1, &padding_temp_1, hidden_dim, batch_size);
    padding_temp_1 /= sizeof(data_t);
    {
        dim3 numBlocks((batch_size - 1) / 32 + 1, (hidden_dim - 1) / 32 + 1);
        dim3 numThreads(32, 32);
        __mul__<<<numBlocks, numThreads>>>(batch_size, input_dim, hidden_dim, x, w_1, temp_1, padding_x, padding_w_1, padding_temp_1);
        // for(int i = 0; i < batch_size; ++i) {
        //     for(int j = 0; j < hidden_dim; ++j) {
        //         temp_1[i * hidden_dim + j] = 0;
        //         for(int k = 0; k < input_dim; ++k) {
        //             temp_1[i * hidden_dim + j] += x[i * input_dim + k] * w_1[k * hidden_dim + j];
        //         }
        //     }
        // }
    }
    // add bias
    {
        dim3 numBlocks((batch_size - 1) / 32 + 1, (hidden_dim - 1) / 32 + 1);
        dim3 numThreads(32, 32);
        __add__<<<numBlocks, numThreads>>>(batch_size, hidden_dim, bias_1, temp_1, padding_temp_1);
        // for(int i = 0; i < batch_size; ++i) {
        //     for(int j = 0; j < hidden_dim; ++j) {
        //         temp_1[i * hidden_dim + j] += bias_1[j];
        //     }
        // }
    }
    // sigmoid
    {
        dim3 numBlocks((batch_size - 1) / 32 + 1, (hidden_dim - 1) / 32 + 1);
        dim3 numThreads(32, 32);
        __sigmoid__<<<numBlocks, numThreads>>>(batch_size, hidden_dim, temp_1, grad_sigmoid_1, padding_temp_1);
        // for(int i = 0; i < batch_size; ++i) {
        //     for(int  j = 0; j < hidden_dim; ++j) {
        //         temp_1[i * hidden_dim + j] = sigmoid(temp_1[i * hidden_dim + j]);
        //         grad_sigmoid_1[j] += temp_1[i * hidden_dim + j] * (1.0 - temp_1[i * hidden_dim + j]);
        //     }
        // }
    }
    // sum up for later calculation
    {
        int numBlocks = (input_dim - 1) / 32 + 1; 
        __accumulate__<<<numBlocks, 32>>>(batch_size, input_dim, x, sum_1, padding_x);
        // for(int i = 0; i < batch_size; ++i) {
        //     for(int j = 0; j < input_dim; ++j) {
        //         sum_1[j] += x[i * input_dim + j];
        //     }
        // }
    }
    // second layer forward
    data_t *temp_2;
    size_t padding_temp_2;
    hipMallocPitch(&temp_2, &padding_temp_2, output_dim * sizeof(data_t), batch_size);
    padding_temp_2 /= sizeof(data_t);
    {   
        dim3 numBlocks((batch_size - 1) / 32 + 1, (output_dim - 1) / 32 + 1);
        dim3 numThreads(32, 32);
        __mul__<<<numBlocks, numThreads>>>(batch_size, hidden_dim, output_dim, temp_1, w_2, temp_2, padding_temp_1, padding_w_2, padding_temp_2); 
        // for(int i = 0; i < batch_size; ++i) {
        //     for(int j = 0; j < output_dim; ++j) {
        //         temp_2[i * output_dim + j] = 0.0;
        //         for(int k = 0; k < hidden_dim; ++k) {
        //             temp_2[i * output_dim + j] += temp_1[i * hidden_dim + k] * w_2[k * output_dim + j];
        //         }
        //     }
        // }
    }
    // add bias
    {
        dim3 numBlocks((batch_size - 1) / 32 + 1, (output_dim - 1) / 32 + 1);
        dim3 numThreads(32, 32);
        __add__<<<numBlocks, numThreads>>>(batch_size, output_dim, bias_2, temp_2, padding_temp_2);
        // for(int i = 0; i < batch_size; ++i) {
        //     for(int j = 0; j < output_dim; ++j) {
        //         temp_2[i * output_dim + j] += bias_2[j];
        //     }
        // }
    }
    // sigmoid
    {
        dim3 numBlocks((batch_size - 1) / 32 + 1, (output_dim- 1) / 32 + 1);
        dim3 numThreads(32, 32);
        __sigmoid__<<<numBlocks, numThreads>>>(batch_size, output_dim, temp_2, grad_sigmoid_2, padding_temp_2);
        // for(int i = 0; i < batch_size; ++i) {
        //     for(int j = 0; j < output_dim; ++j) {
        //         temp_2[i * output_dim + j] = sigmoid(temp_2[i * output_dim + j]);
        //         grad_sigmoid_2[j] += temp_2[i * output_dim + j] * (1.0 - temp_2[i * output_dim + j]);
        //     }
        // }
    }
    // sum up for later calculation
    {
        int numBlocks = (hidden_dim - 1) / 32 + 1; 
        __accumulate__<<<numBlocks, 32>>>(batch_size, hidden_dim, temp_1, sum_2, padding_temp_1);
        // for(int i = 0; i < batch_size; ++i) {
        //     for(int j = 0; j < hidden_dim; ++j) {
        //         sum_2[j] += temp_1[i * hidden_dim + j];
        //     }
        // }
    }
    *padding_pred = padding_temp_2;
    hipFree(temp_1);
    return temp_2;
}

__global__ void __loss__(const size_t batch_size, data_t *pred, data_t *real, data_t *grad, data_t *err, size_t padding_pred, size_t padding_real) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < batch_size) {
        data_t temp_err = 0;
        for(int j = 0; j < output_dim; ++j) {
            data_t temp = pred[i * padding_pred + j] - real[i * padding_real + j];
            atomicAdd(grad + j, temp);
            temp_err += temp * temp;
        }
        atomicAdd(err, temp_err);
    }
}

data_t model::loss(size_t batch_size, data_t *pred, data_t *real, size_t padding_pred, size_t padding_real) {
    data_t err = 0.0;
    data_t *device_err;
    hipMalloc(&device_err, sizeof(data_t));
    hipMemset(device_err, 0, sizeof(data_t));
    int numBlocks = (batch_size - 1) / 32 + 1;
    __loss__<<<numBlocks, 32>>>(batch_size, pred, real, grad_loss, device_err, padding_pred, padding_real);
    hipMemcpy(&err, device_err, sizeof(data_t), hipMemcpyDeviceToHost);
    hipFree(device_err); 
    hipFree(pred);
    return err / (data_t)(batch_size * output_dim);
    // data_t temp, err = 0.0;
    // for(int i = 0; i < batch_size; ++i) {
    //     for(int j = 0; j < output_dim; ++j) {
    //         temp = pred[i * output_dim + j] - real[i * output_dim + j];
    //         grad_loss[j] += temp;
    //         err += temp * temp;
    //     }
    // }
    // return err / (data_t)(batch_size * output_dim);
}

void model::zero_grad() {
    batch_count = 0;
    fill_val(input_dim, sum_1, 0);
    fill_val(hidden_dim, sum_2, 0);
    fill_val(hidden_dim, grad_sigmoid_1, 0);
    fill_val(output_dim, grad_sigmoid_2, 0);
    fill_val(hidden_dim, grad_bias_1, 0);
    fill_val(output_dim, grad_bias_2, 0);
    fill_val(output_dim, grad_loss, 0);
    fill_val(hidden_dim, prev_grad, 0);
    fill_val2D(input_dim, hidden_dim, grad_w_1, 0, padding_grad_w_1);
    fill_val2D(hidden_dim, output_dim, grad_w_2, 0, padding_grad_w_2);
}

void model::backward() {
    // grad of loss
    {
        int numBlocks = (output_dim - 1) / 32 + 1;
        __div__<<<numBlocks, 32>>>(output_dim, grad_loss, batch_count * output_dim / 2.0);
        // for(int i = 0; i < output_dim; ++i) {
        //     grad_loss[i] /= batch_count * output_dim / 2.0;
        // }
    }
    // grad of second sigmoid
    {
        int numBlocks = (output_dim - 1) / 32 + 1;
        __div__<<<numBlocks, 32>>>(output_dim, grad_sigmoid_2, batch_count);
        __mul__<<<numBlocks, 32>>>(output_dim, grad_sigmoid_2, grad_loss, grad_sigmoid_2);
        // for(int i = 0; i < output_dim; ++i) {
        //     grad_sigmoid_2[i] /= batch_count;
        //     grad_sigmoid_2[i] *= grad_loss[i];
        // }
        
    }
    // grad of second bias
    {
        int numBlocks = (output_dim - 1) / 32 + 1;
        __copy__<<<numBlocks, 32>>>(output_dim, grad_sigmoid_2, grad_bias_2);
        // for(int i = 0; i < output_dim; ++i) {
        //     grad_bias_2[i] = grad_sigmoid_2[i];
        // }
    }
    // grad of second w
    {
        int numBlocks = (hidden_dim - 1) / 32 + 1;
        __div__<<<numBlocks, 32>>>(hidden_dim, sum_2, batch_count);
        // for(int i = 0; i < hidden_dim; ++i) {
        //     sum_2[i] /= batch_count;
        // }
    }
    {
        dim3 numBlocks((hidden_dim - 1) / 32 + 1, (output_dim - 1) / 32 + 1);
        dim3 numThreads(32, 32);
        __dot__<<<numBlocks, numThreads>>>(hidden_dim, output_dim, sum_2, grad_sigmoid_2, grad_w_2, padding_grad_w_2);
        // for(int i = 0; i < hidden_dim; ++i) {
        //     for(int j = 0; j < output_dim; ++j) {
        //         grad_w_2[i * output_dim + j] = sum_2[i] * grad_sigmoid_2[j];
        //     }
        // }
    }
    {
        int numBlocks = (hidden_dim - 1) / 32 + 1;
        __sum__<<<numBlocks, 32>>>(hidden_dim, output_dim, w_2, grad_sigmoid_2, prev_grad, padding_w_2);
        // for(int i = 0; i < hidden_dim; ++i) {
        //     prev_grad[i] = 0.0;
        //     for(int j = 0; j < output_dim; ++j) {
        //         prev_grad[i] += w_2[i * output_dim + j] * grad_sigmoid_2[j];
        //     }
        // }
    }
    
    // grad of first sigmoid
    {
        int numBlocks = (hidden_dim - 1) / 32 + 1;
        __div__<<<numBlocks, 32>>>(hidden_dim, grad_sigmoid_1, batch_count);
        __mul__<<<numBlocks, 32>>>(hidden_dim, grad_sigmoid_1, prev_grad, grad_sigmoid_1);
        // for(int i = 0; i < hidden_dim; ++i) {
        //     grad_sigmoid_1[i] /= batch_count;
        //     grad_sigmoid_1[i] *= prev_grad[i];
        // }
    }
    // grad of first bias
    {
        int numBlocks = (hidden_dim - 1) / 32 + 1;
        __copy__<<<numBlocks, 32>>>(hidden_dim, grad_sigmoid_1, grad_bias_1);
        // for(int i = 0; i < hidden_dim; ++i) {
        //     grad_bias_1[i] = grad_sigmoid_1[i];
        // }
    }
    // grad of first w
    {
        int numBlocks = (input_dim - 1) / 32 + 1;
        __div__<<<numBlocks, 32>>>(input_dim, sum_1, batch_count);
        // for(int i = 0; i < input_dim; ++i) {
        //     sum_1[i] /= batch_count;
        // }
    }
    {   
        dim3 numBlocks((input_dim - 1) / 32 + 1, (hidden_dim - 1) / 32 + 1);
        dim3 numThreads(32, 32);
        __dot__<<<numBlocks, numThreads>>>(input_dim, hidden_dim, sum_1, grad_sigmoid_1, grad_w_1, padding_grad_w_1);
        // for(int i = 0; i < input_dim; ++i) {
        //     for(int j = 0; j < hidden_dim; ++j) {
        //         grad_w_1[i * hidden_dim + j] = sum_1[i] * grad_sigmoid_1[j];
        //     }
        // }
    }
}

void model::update(data_t lr) {
    // update linear layer 1
    {
        dim3 numBlocks((input_dim - 1) / 32 + 1, (hidden_dim - 1) / 32 + 1);
        dim3 numThreads(32, 32);
        __sub_eq_2D__<<<numBlocks, numThreads>>>(input_dim, hidden_dim, grad_w_1, w_1, lr, padding_grad_w_1, padding_w_1);
        // for(int i = 0; i < input_dim; ++i) {
        //     for(int j = 0; j < hidden_dim; ++j) {
        //         w_1[i * hidden_dim + j] -= lr * grad_w_1[i * hidden_dim + j];
        //     }
        // }
    }
    {
        int numBlocks = (hidden_dim - 1) / 32 + 1;
        __sub_eq__<<<numBlocks, 32>>>(hidden_dim, grad_bias_1, bias_1, lr);
        // for(int i = 0; i < hidden_dim; ++i) { 
        //     bias_1[i] -= lr * grad_bias_1[i];
        // }
    }    
    // update linear layer 2
    {
        dim3 numBlocks((input_dim - 1) / 32 + 1, (hidden_dim - 1) / 32 + 1);
        dim3 numThreads(32, 32);
        __sub_eq_2D__<<<numBlocks, numThreads>>>(hidden_dim, output_dim, grad_w_2, w_2, lr, padding_grad_w_2, padding_w_2);
        // for(int i = 0; i < hidden_dim; ++i) {
        //     for(int j = 0; j < output_dim; ++j) {
        //         w_2[i * output_dim + j] -= lr * grad_w_2[i * output_dim + j];
        //     }
        // }
    }
    {
        int numBlocks = (output_dim - 1) / 32 + 1;
        __sub_eq__<<<numBlocks, 32>>>(output_dim, grad_bias_2, bias_2, lr);
        // for(int i = 0; i < output_dim; ++i) {
        //     bias_2[i] -= lr * grad_bias_2[i];
        // }
    }
}









